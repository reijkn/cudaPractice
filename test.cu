
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Kernel()
{
    // スレッド番号の取得
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World! thread:%d\n", i);
}

int main(void)
{
    Kernel<<<1, 10>>>();
    hipDeviceSynchronize();

    std::cout << "done!" << std::endl;

    return 0;
}